#include "hip/hip_runtime.h"
// cuda functions to execute collision detection
// and synchrozie data between the detector class 
// and the gpu hosted memory
// reference: https://zhuanlan.zhihu.com/p/34587739

#include "collide.h"
#include <iostream>
#include <cstdio>
#include <glm/glm.hpp>
using namespace std;
using namespace glm;

// define host data
glm::vec3 pos[MAX_BALL_COUNT], velocity[MAX_BALL_COUNT];
float mass[MAX_BALL_COUNT], radius[MAX_BALL_COUNT], cor[MAX_BALL_COUNT];
int b1[MAX_COLLISIONS], b2[MAX_COLLISIONS];
int b[MAX_COLLISIONS], p[MAX_COLLISIONS];

// define device data
__device__ glm::vec3 _pos[MAX_BALL_COUNT], _velocity[MAX_BALL_COUNT];
__device__ float _mass[MAX_BALL_COUNT], _radius[MAX_BALL_COUNT], _cor[MAX_BALL_COUNT];
__device__ int _b1[MAX_COLLISIONS], _b2[MAX_COLLISIONS];
__device__ int _b[MAX_COLLISIONS], _p[MAX_COLLISIONS];

// sychronize data between device and host 
void reverseSyncVelocity(int n) {
	hipMemcpyFromSymbol(velocity, HIP_SYMBOL(_velocity), n * sizeof(vec3));
}

__device__ void printv(vec3 val) {
	printf("%f %f %f\n", val.x, val.y, val.z);
}

__device__ void printfl(float val) {
	printf("%f\n", val);
}

void syncVars(int n) {
	hipMemcpyToSymbol(HIP_SYMBOL(_pos), pos, n * sizeof(vec3), 0);
	hipMemcpyToSymbol(HIP_SYMBOL(_velocity), velocity, n * sizeof(vec3), 0);
}

void syncConsts(int n) {
	hipMemcpyToSymbol(HIP_SYMBOL(_mass), mass, n * sizeof(float), 0);
	hipMemcpyToSymbol(HIP_SYMBOL(_radius), radius, n * sizeof(float), 0);
	hipMemcpyToSymbol(HIP_SYMBOL(_cor), cor, n * sizeof(float), 0);
}

void syncBallPairs(int n) {
	hipMemcpyToSymbol(HIP_SYMBOL(_b1), b1, n * sizeof(int), 0);
	hipMemcpyToSymbol(HIP_SYMBOL(_b2), b2, n * sizeof(int), 0);
}

void syncBallPlanePairs(int n) {
	hipMemcpyToSymbol(HIP_SYMBOL(_b), b, n * sizeof(int), 0);
	hipMemcpyToSymbol(HIP_SYMBOL(_p), p, n * sizeof(int), 0);
}

// synchronization between cuda and detector class
void initBallCuda(vector<Ball*> balls, int n) {
    for (int i = 0; i < n; i++) {
        pos[i] = balls[i]->pos;
        velocity[i] = balls[i]->velocity;
        mass[i] = balls[i]->mass;
        cor[i] = balls[i]->cor;
        radius[i] = balls[i]->radius;
    }
	syncConsts(n);
	syncVars(n);
}

void copyBallVarCuda(vector<Ball*> balls, int n) {
    for (int i = 0; i < n; i++) {
        pos[i] = balls[i]->pos;
        velocity[i] = balls[i]->velocity;
    }
	syncVars(n);
	return;
}

void updateVelocityCuda(vector<Ball*> balls, int n) {
	reverseSyncVelocity(n);
    for (int i = 0; i < n; i++) {
		balls[i]->velocity = velocity[i];
    }
}

void copyBallPairCuda(vector<BallPair> pairs, int numPairs) {
    for (int i = 0; i < numPairs && i < MAX_COLLISIONS; i++) {
        b1[i] = pairs[i].b1;
        b2[i] = pairs[i].b2;
    }
	syncBallPairs(numPairs);
}

void copyBallPlanePairCuda(vector<BallPlanePair> pairs, int numPairs) {
    for (int i = 0; i < numPairs && i < MAX_COLLISIONS; i++) {
        b[i] = pairs[i].b;
        p[i] = pairs[i].p;
    }
	syncBallPlanePairs(numPairs);
}

// utility function to compute the plane normal vector
__device__
vec3 _planeDir(int p) {
	switch (p) {
	case LEFT:
		return vec3(-1.0f, 0.0f, 0.0f);
		break;
	case RIGHT:
		return vec3(1.0f, 0.0f, 0.0f);
		break;
	case BACK:
		return vec3(0.0f, 0.0f, -1.0f);
		break;
	case FRONT:
		return vec3(0.0f, 0.0f, 1.0f);
		break;
	case TOP:
		return vec3(0.0f, 1.0f, 0.0f);
		break;
	case BOTTOM:
		return vec3(0.0f, -1.0f, 0.0f);
		break;
	default:
		return vec3(0.0f, 0.0f, 0.0f);
	}
}

// kernel functions
__global__
void ballCollideKernel(int numPairs) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < numPairs; i += stride) {
        int index1 = _b1[i];
        int index2 = _b2[i];
		vec3 pos1 = _pos[index1];
		vec3 pos2 = _pos[index2];
        float r = _radius[index1] + _radius[index2];
		vec3 dp = pos1 - pos2;
		vec3 v1 = _velocity[index1];
		vec3 v2 = _velocity[index2];
		vec3 dv = v1 - v2;
        if (dot(dp, dp) < r * r && dot(dv, dp) <= 0) {
			// balls are close enough and are approaching
			// so the collision will happen
			float cor1 = _cor[index1];
			float cor2 = _cor[index2];
			float c = min(cor1, cor2);
			float m1 = _mass[index1];
			float m2 = _mass[index2];

			// use momentum & energy preservation theorem
			// to solve the velocities
			vec3 dpvec = normalize(dp);
			vec3 proj1 = dot(v1, dpvec) * dpvec;
			vec3 proj2 = dot(v2, dpvec) * dpvec;
			vec3 dv1 = ((1 + c) * m2 * (proj2 - proj1)) / (m1 + m2);
			vec3 dv2 = ((1 + c) * m1 * (proj1 - proj2)) / (m1 + m2);
			_velocity[index1] += dv1;
			_velocity[index2] += dv2;
        }
    }
}

__global__
void ballPlaneCollideKernel(int numPairs) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < numPairs; i += stride) {
        int ballIndex = _b[i];
        int planeIndex = _p[i];
		vec3 dir = _planeDir(planeIndex);
		vec3 p = _pos[ballIndex];
		vec3 v = _velocity[ballIndex];
        float r = _radius[ballIndex];
        if (dot(p, dir) + r > SIZE / 2 && dot(v, dir) > 0) {
			// the ball and the plane are close enough and the ball is approaching
			// so the collision will happen
            float c = _cor[ballIndex];

			// assume the plane is of infinity mass
			vec3 dv = (1 + c) * dir * dot(v, dir);
			_velocity[ballIndex] -= dv;
        }
    }
}

// interfaces to the detector
void ballCollideCuda(vector<BallPair>& pairs, vector<Ball*> balls) {
	int numBalls = balls.size();
	int numPairs = pairs.size();
	copyBallPairCuda(pairs, numPairs);

	dim3 blockSize(64);
	dim3 gridSize((numBalls + blockSize.x - 1) / blockSize.x);

	// call kernel function
	ballCollideKernel <<<gridSize, blockSize>>> (numPairs);
}

void ballPlaneCollideCuda(vector<BallPlanePair>& pairs, vector<Ball*> balls) {
	int numBalls = balls.size();
	int numPairs = pairs.size();
	copyBallPlanePairCuda(pairs, numPairs);

	dim3 blockSize(64);
	dim3 gridSize((numBalls + blockSize.x - 1) / blockSize.x);

	// call kernel function
	ballPlaneCollideKernel <<<gridSize, blockSize>>> (numPairs);
	auto result = hipGetLastError();
}

